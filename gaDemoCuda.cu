#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <malloc.h>

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 100
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize


//typedef float float;
float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];
// float Dzsum[9];
// float populationArray[populationSize][chromosomeSize];  //种群数组
// float fitness[populationSize]; //每个种群的适应度
// float populationPro[populationSize]; //每个种群在select时被选中的概率
float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度
float X_10[chromosomeSize]; //最优适应度对应的x值
float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数
// BOOL elitism = TRUE; //是否精英选择

float *createMatrix(int rows, int cols) {
    float *matrix = (float*)malloc(rows * cols * sizeof(float));
    return matrix;
}

//Get data from files
BOOL getData(const char *fileName, float *x, int rows, int cols) {
    // open file to read data
    FILE *fp;
    fp = fopen(fileName, "r");
    if (fp == NULL) {
        printf("Open file %s error!!\n", fileName);
        return FALSE;
    }

    // read data
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fscanf(fp, "%f", (x + i * cols + j));
        }
    }
    return TRUE;
}

// //sig2ext in rainflow
__host__ __device__ float *sig2ext(float *sigy, float *dty, int rows, int *lenOfArray) {

    int n = 0, m = 0, k = 0, l = 0;

    //w=diff(sig);
    //w=logical([1;(w(1:end-1).*w(2:end))<=0;1]);
    float *w = (float*)malloc(rows * sizeof(float));
    for (int i = 1; i < rows; i++) {
        w[i - 1] = sigy[i] - sigy[i - 1];
    }

    for (int i = rows - 2; i > 0; i--) {
        float tmp = w[i] * w[i - 1];
        if (tmp <= 0) {
            w[i] = 1;
        } else {
            w[i] = 0;
            n++;
        }
    }
    w[0] = w[rows - 1] = 1;

    //ext=sigy(w);  exttime=dty(w);
    float *ext = (float*)malloc((rows - n) * sizeof(float));
    float *exttime = (float*)malloc((rows - n) * sizeof(float));
    for (int i = 0, j = 0; i < rows - n && j < rows;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = sigy[j];
            exttime[i] = dty[j];
            i++;
            j++;
        }
    }

    //w=diff(ext);
    //w=~logical([0; w(1:end-1)==0 & w(2:end)==0; 0]);
    for (int i = 1; i < rows - n; i++) {
        w[i - 1] = ext[i] - ext[i - 1];
    }

    for (int i = rows - n - 2; i > 0; i--) {
        if (w[i - 1] == 0 && w[i] == 0) {
            w[i] = 0;
            m++;
        } else {
            w[i] = 1;
        }
    }
    w[0] = w[rows - n - 1] = 1;

    //ext=ext(w); exttime=exttime(w);
    for (int i = 0, j = 0; i < rows - n - m && j < rows - n;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = ext[j];
            exttime[i] = exttime[j];
            i++;
            j++;
        }
    }

    //w=~logical([0; ext(1:end-1)==ext(2:end)]);
    for (int i = 1; i < rows - n - m; i++) {
        if (ext[i - 1] == ext[i]) {
            w[i] = 0;
            k++;
        } else {
            w[i] = 1;
        }
    }
    w[0] = 1;

    //ext=ext(w);
    for (int i = 0, j = 0; i < rows - n - m - k && j < rows - n - m;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = ext[j];
            i++;
            j++;
        }
    }
    //w2=(exttime(2:end)-exttime(1:end-1))./2
    //exttime=[exttime(1:end-1)+w2.*~w(2:end); exttime(end)];
    //exttime=exttime(w);

    float *w2 = (float*)malloc((rows - n - m - 1) * sizeof(float));
    for (int i = 1; i < rows - n - m; i++) {
        w2[i - 1] = (exttime[i] - exttime[i - 1]) / 2.00;
    }

    for (int i = 0, j = 1; i < rows - n - m - 1 && j < rows - n - m;) {
        if (w[j] == 0) {
            exttime[i] = w2[i] * 1.00 + exttime[i];
            i++;
            j++;
        } else {
            exttime[i] = w2[i] * 0.00 + exttime[i];
            i++;
            j++;
        }
    }

    for (int i = 0, j = 0; i < rows - n - m - k && j < rows - n - m;) {
        if (w[j] == 0) {
            j++;
        } else {
            exttime[i] = exttime[j];
            i++;
            j++;
        }
    }

    //length(ext)>2,  w=diff(ext); w=logical([1; w(1:end-1).*w(2:end)<0; 1]);
    //ext4=ext(w); exttime=exttime(w);
    float *ext4 = NULL;
    *lenOfArray = 0;
    if (rows - n - m - k > 2) {
        for (int i = 1; i < rows - n - m - k; i++) {
            w[i - 1] = ext[i] - ext[i - 1];
        }

        for (int i = rows - n - m - k - 2; i > 0; i--) {
            if (w[i - 1] * w[i] < 0) {
                w[i] = 1;
            } else {
                w[i] = 0;
                l++;
            }
        }
        w[0] = 1;
        w[rows - n - m - k - 1] = 1;

        *lenOfArray = rows - n - m - k - l;
        ext4 = (float*)malloc(2 * (*lenOfArray) * sizeof(float));

        for (int i = 0, j = 0; i < rows - n - m - k - l && j < rows - n - m - k;) {
            if (w[j] == 0) {
                j++;
            } else {
                ext4[i] = ext[j];
                ext4[i + (*lenOfArray)] = exttime[j];
                i++;
                j++;
            }
        }
    }

    free(w);
    free(w2);
    free(ext);
    free(exttime);
    return ext4;
}

// //rainFlow in rainflow
__host__ __device__ float *rainFlow(float *ext, float *exttime, int lenOfSig2ext, int *lenOfRainflow) {
    float *rfy = NULL, *rfyResult = NULL;

    //function rfy5
    float a[100], t[100], ampl, mean, period, atime;
    int cNr = 1;
    int j = -1;

    //create 2D rfy(5 * (lenOfSig2ext -1))
    rfy = (float*)malloc(5 * (lenOfSig2ext - 1) * sizeof(float));

    int columnId = 0;
    int pointId = 0;

    for (int i = 0; i < lenOfSig2ext; i++) {
        a[++j] = *(ext + pointId);
        t[j] = *(exttime + pointId);
        while ((j >= 2) && (fabs(a[j - 1] - a[j - 2]) <= fabs(a[j] - a[j - 1]))) {
            ampl = fabs((a[j - 1] - a[j - 2]) / 2);
            switch (j) {
                case 0: {
                    break;
                }
                case 1: {
                    break;
                }
                case 2: {
                    mean = (a[0] + a[1]) / 2;
                    period = (t[1] - t[0]) * 2;
                    atime = t[0];
                    a[0] = a[1];
                    a[1] = a[2];
                    t[0] = t[1];
                    t[1] = t[2];
                    j = 1;
                    if (ampl > 0) {
                        *(rfy + columnId*5 + 0) = ampl;
                        *(rfy + columnId*5 + 1) = mean;
                        *(rfy + columnId*5 + 2) = 0.50;
                        *(rfy + columnId*5 + 3) = atime;
                        *(rfy + columnId*5 + 4) = period;
                        columnId++;
                    }
                    break;
                }
                default: {
                    mean = (a[j - 1] + a[j - 2]) / 2;
                    period = (t[j - 1] - t[j - 2]) * 2;
                    atime = t[j - 2];
                    a[j - 2] = a[j];
                    t[j - 2] = t[j];
                    j = j - 2;
                    if (ampl > 0) {
                        *(rfy + columnId*5 + 0) = ampl;
                        *(rfy + columnId*5 + 1) = mean;
                        *(rfy + columnId*5 + 2) = 1.00;
                        *(rfy + columnId*5 + 3) = atime;
                        *(rfy + columnId*5 + 4) = period;
                        columnId++;
                        cNr++;
                    }
                    break;
                }
            }
        }
        pointId++;
    }
    for (int i = 0; i < j; i++) {
        ampl = fabs(a[i] - a[i + 1]) / 2;
        mean = (a[i] + a[i + 1]) / 2;
        period = (t[i + 1] - t[i]) * 2;
        atime = t[i];
        if (ampl > 0) {
            *(rfy + columnId*5 + 0) = ampl;
            *(rfy + columnId*5 + 1) = mean;
            *(rfy + columnId*5 + 2) = 0.50;
            *(rfy + columnId*5 + 3) = atime;
            *(rfy + columnId*5 + 4) = period;
            columnId++;
        }
    }

    //create 2D rfyResult(5 * (lenOfSig2ext - cNr))
    rfyResult = (float*)malloc(5 * (lenOfSig2ext - cNr) * sizeof(float));

    *lenOfRainflow = lenOfSig2ext - cNr;

    for (int i = 0; i < 5 * (lenOfSig2ext - cNr); i++) {
            rfyResult[i] = rfy[i];
    }

    free(rfy);
    return rfyResult;
}

// //rfhist in rainflow
__host__ __device__ float *rfhist(float *rfy, int lenOfRainflow, int *lenOfRfhist) {
    float *noy = NULL, *xoy = NULL;
    int x = 32;
    *lenOfRfhist = x;

    xoy = (float*)malloc(x * sizeof(float));
    noy = (float*)malloc(x * sizeof(float));
    memset(noy, 0, x * sizeof(float));

    //halfc=find(rfy(3,:)==0.5)
    int halfcNum = 0;
    for (int i = 0; i < lenOfRainflow; i++) {
        if (rfy[i * 5 + 2] == 0.50)
            halfcNum++;
    }

    int *halfc = NULL;
    halfc = (int*)malloc(halfcNum * sizeof(int));
    for (int i = 0, j = 0; i < lenOfRainflow && j < halfcNum;) {
        if (rfy[i * 5 + 2] == 0.50) {
            halfc[j] = i;
            j++;
        }
        i++;
    }

    float min = rfy[0], max = rfy[0];
    for (int i = 0; i < lenOfRainflow; i++) {
        if (rfy[i * 5] >= max) {
            max = rfy[i * 5];
        } else if (rfy[i * 5] <= min) {
            min = rfy[i * 5];
        }
    }

    float wid = (max - min) / x;
    for (int i = 0; i < x; i++) {
        xoy[i] = min + (float) (i + 0.50) * wid;
    }

    for (int i = 0; i < lenOfRainflow; i++) {
        int j;
        j = (int) floor((rfy[i * 5] - min) / wid);
        if (j != 0 && fabs((rfy[i * 5] - min) - wid * j) < 1e-10) {
            noy[j - 1] += 1;
        } else {
            noy[j] += 1;
        }
    }

    //if ~isempty(halfc) {
    //[N2 x]=hist(rf(r,halfc),x)  N2 = noy2, x = *xoy
    //N1=N1-0.5*N2  N1 = noy
    // }
    if (halfcNum != 0) {
        float *noy2 = (float*)malloc(x * sizeof(float));
        memset(noy2, 0, x * sizeof(float));
        float *rf = (float*)malloc(halfcNum * sizeof(float));
        for (int i = 0; i < halfcNum; i++) {
            int j = halfc[i];
            rf[i] = rfy[j * 5];
        }

        for (int i = 0; i < halfcNum; i++) {
            int j;
            j = (int) floor((rf[i] - min) / wid);
            if (j != 0 && fabs((rf[i] - min) - wid * j) < 1e-10) {
                noy2[j - 1] += 1;
            } else {
                noy2[j] += 1;
            }
        }

        for (int i = 0; i < x; i++) {
            noy[i] -= noy2[i] * 0.5;
        }

        free(noy2);
        free(rf);
    }

    float *rfhist = (float*)malloc(2 * x * sizeof(float));
    for (int i = 0; i < x; i++) {
        rfhist[i] = noy[i];
        rfhist[i + x] = xoy[i];
    }

    free(halfc);
    free(xoy);
    free(noy);

    return rfhist;
}

// __global__ void c_testPreData(float *c_aaa, int aaaRow, int aaaCol, float *c_Dysum) {
//     memset(c_Dysum, 0, sizeof(float) * 9);
    
//     int idx = threadIdx.x;
//     float *sigy = (float*)malloc(aaaRow * sizeof(float));
//     float *dty = (float*)malloc(aaaRow * sizeof(float));

//     for (int i = 0; i < aaaRow; i++) {
//         sigy[i] = c_aaa[i * aaaCol + idx + 2];
//         dty[i] = c_aaa[i * aaaCol + 1];
//     }
//     __syncthreads();

//     float *ext = NULL, *exttime = NULL;
//     int lenOfSig2ext;
//     ext = sig2ext(sigy, dty, aaaRow, &lenOfSig2ext);
//     //__syncthreads();
//     exttime = ext + lenOfSig2ext;
//     __syncthreads();

//     float *rfy = NULL;
//     int lenOfRainflow;
//     rfy = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);
//     __syncthreads();

//     float *noy = NULL, *xoy = NULL;
//     int lenOfRfhist;
//     noy = rfhist(rfy, lenOfRainflow, &lenOfRfhist);
//     //__syncthreads();
//     xoy = noy + lenOfRfhist;
//     __syncthreads();

//     for (int i = 0; i < lenOfRfhist; i++) {
//         c_Dysum[idx] += noy[i] * pow(xoy[i] * 0.21 / 70, 3.5);
//     }
//     __syncthreads();
//     printf("%e\n", c_Dysum[idx]);

//     free(sigy);
//     free(dty);
//     free(ext);
//     free(rfy);
//     free(noy);
// }

void testPreData() {
    for (int kk = 0; kk < 9; kk++) {
        float *sigy = (float*)malloc(aaaRow * sizeof(float));
        float *dty = (float*)malloc(aaaRow * sizeof(float));

        for (int i = 0; i < aaaRow; i++) {
            sigy[i] = aaa[i * 11 + kk + 2];
            dty[i] = aaa[i * 11 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sigy, dty, aaaRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rfy = NULL;
        int lenOfRainflow;
        rfy = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *noy = NULL, *xoy = NULL;
        int lenOfRfhist;
        noy = rfhist(rfy, lenOfRainflow, &lenOfRfhist);
        xoy = noy + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dysum[kk] += noy[i] * pow(xoy[i] * 0.21 / 70, 3.5);
        }
        //printf("%e\n", Dysum[kk]);

        free(sigy);
        free(dty);
        free(ext);
        free(rfy);
        free(noy);
    }
}


// __device__ float sum(float *x) {
//     float sum = 0;
//     for (int i = 0; i < populationSize; i++) {
//         sum += x[i];
//     }
//     return sum;
// }

 //fitness Function
float HfitnessFcn(float *x) {
    //initial Dzsum in every generation
    float *Dzsum = (float*)malloc(9 * sizeof(float));
    memset(Dzsum, 0, sizeof(float) * 9);

    float *Tzb = (float*)malloc(aRow * 9 * sizeof(float));

    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < aRow; j++) {
            Tzb[j * 9 + i] = x[0] * aa[0 * 9 + i] * a[j * 16 + 2] + x[1] * aa[1 * 9 + i] * a[j * 16 + 3] + x[2] * aa[2 *0 + i] * a[j * 16 + 4] +
                            x[3] * aa[3 * 9 + i] * a[j * 16 + 5] + x[4] * aa[4 * 9 + i] * a[j * 16 + 6] + x[5] * aa[5 * 9 + i] * a[j * 16 + 7] +
                            x[6] * aa[6 * 9 + i] * a[j * 16 + 8] + x[7] * aa[7 * 9 + i] * a[j * 16 + 9] + x[8] * aa[8 * 9 + i] * a[j * 16 + 10] +
                            x[9] * aa[9 * 9 + i] * a[j * 16 + 11];
        }
    }

    for (int k = 0; k < 9; k++) {
        float *sig = (float*)malloc(aRow * sizeof(float));
        float *dt = (float*)malloc(aRow * sizeof(float));
        for (int i = 0; i < aRow; i++) {
            sig[i] = Tzb[i * 9 + k];
            dt[i] = a[i * 9 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sig, dt, aRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rf = NULL;
        int lenOfRainflow;
        rf = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *no = NULL, *xo = NULL;
        int lenOfRfhist;
        no = rfhist(rf, lenOfRainflow, &lenOfRfhist);
        xo = no + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dzsum[k] += no[i] * pow(xo[i] * 0.21 / 70, 3.5);
        }
        //printf("%e\n", Dzsum[k]);

        free(sig);
        free(dt);
        free(ext);
        free(rf);
        free(no);
    }

    float y = 0;
    for (int i = 0; i < 9; i++) {
        //constraint : c =(Dysum[i]-Dzsum[i]) <= 0
        float c = Dysum[i] - Dzsum[i];
        if (c <= 0) {
            y += pow(c, 2);
        } else {
            y = 100;
        }
    }
    //printf("%e\n", y);

    free(Dzsum);
    free(Tzb);

    return y;
}

__device__ float DfitnessFcn(float *x) {
    //initial Dzsum in every generation
    float *Dzsum = (float*)malloc(9 * sizeof(float));
    memset(Dzsum, 0, sizeof(float) * 9);

    float *Tzb = (float*)malloc(c_aRow * 9 * sizeof(float));

    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < c_aRow; j++) {
            Tzb[j * 9 + i] = x[0] * c_aa[0 * 9 + i] * c_a[j * 16 + 2] + x[1] * c_aa[1 * 9 + i] * c_a[j * 16 + 3] + x[2] * c_aa[2 *0 + i] * c_a[j * 16 + 4] +
                            x[3] * c_aa[3 * 9 + i] * c_a[j * 16 + 5] + x[4] * c_aa[4 * 9 + i] * c_a[j * 16 + 6] + x[5] * c_aa[5 * 9 + i] * c_a[j * 16 + 7] +
                            x[6] * c_aa[6 * 9 + i] * c_a[j * 16 + 8] + x[7] * c_aa[7 * 9 + i] * c_a[j * 16 + 9] + x[8] * c_aa[8 * 9 + i] * c_a[j * 16 + 10] +
                            x[9] * c_aa[9 * 9 + i] * c_a[j * 16 + 11];
        }
    }

    for (int k = 0; k < 9; k++) {
        float *sig = (float*)malloc(c_aRow * sizeof(float));
        float *dt = (float*)malloc(c_aRow * sizeof(float));
        for (int i = 0; i < c_aRow; i++) {
            sig[i] = Tzb[i * 9 + k];
            dt[i] = c_a[i * 9 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sig, dt, c_aRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rf = NULL;
        int lenOfRainflow;
        rf = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *no = NULL, *xo = NULL;
        int lenOfRfhist;
        no = rfhist(rf, lenOfRainflow, &lenOfRfhist);
        xo = no + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dzsum[k] += no[i] * pow(xo[i] * 0.21 / 70, 3.5);
        }
        printf("%e\n", Dzsum[k]);

        free(sig);
        free(dt);
        free(ext);
        free(rf);
        free(no);
    }

    float y = 0;
    for (int i = 0; i < 9; i++) {
        //constraint : c =(Dysum[i]-Dzsum[i]) <= 0
        float c = c_Dysum[i] - Dzsum[i];
        if (c <= 0) {
            y += pow(c, 2);
        } else {
            y = 100;
        }
    }
    printf("%e\n", y);

    free(Dzsum);
    //free(Tzb);

    return y;
}


__global__ void GfitnessFcn(float *populationArray, float *fitness){
    int idx = threadIdx.x;
    float *x = (float*)malloc(chromosomeSize * sizeof(float));
    memset(x, 0, 10 * sizeof(float));
    for (int j = 0; j < chromosomeSize; j++) {
        x[j] = populationArray[idx * chromosomeSize + j];
        //printf("%f\n", x[j]);
    }
    __syncthreads();
    fitness[idx] = DfitnessFcn(Tzb);
    free(x);
}



void initial(float *populationArray){
    for (int i = 0; i < populationSize; i++) {
        float *x = (float*)malloc(chromosomeSize * sizeof(float));
        for (int j = 0; j < chromosomeSize; j++) {
            int high_pos = rand();
            int low_pos = (rand() & ((1 << 16) - 1));
            high_pos = (high_pos & ((1 << 15) - 1));
            int value = low_pos + (high_pos << 16);
            populationArray[i * chromosomeSize + j] = (UB[j] - LB[j]) * ((float) value / ((1U << 31) - 1)) + LB[j];
            x[j] = populationArray[i * chromosomeSize + j];
        }
        float tmp_fit = HfitnessFcn(x);
        if (tmp_fit > 99) {
            i--;
        }
        free(x);
    }
}

// __device__ void *bestFitness() {

//     //bestRes[bestFitness][bestIndex]
//     float c_bestFitness = c_fitness[0];
//     int c_bestIndex = 0;
//     float *bestRes = (float*)malloc(2 * sizeof(float));
//     for (int i = 0; i < populationSize; i++) {
//         if (c_fitness[i] < c_bestFitness) {
//             c_bestFitness = c_fitness[i];
//             c_bestIndex = i;
//         }
//     }
//     bestRes[0] = c_bestFitness;
//     bestRes[1] = c_bestIndex;

//     return bestRes;
// }

// //select function 轮盘选择
// __device__ void selectFcn() {

//     float tmpPopulationArray[populationSize][chromosomeSize];
//     float tmpFitness[populationSize];
//     //每个个体被选择的概率
//     float *Fitness = malloc(populationSize * sizeof(float));
//     float sumFitness = 0;

//     for (int i = 0; i < populationSize; i++) {
//         Fitness[i] = 1 / fitness[i];
//     }

//     sumFitness = sum(Fitness);
//     for (int i = 0; i < populationSize; i++) {
//         populationPro[i] = Fitness[i] / sumFitness;
//     }
//     free(Fitness);

//     //轮盘选择
//     int *index = malloc(populationSize * sizeof(int));
//     for (int i = 0; i < populationSize; i++) {
//         float pick = ((float) rand()) / RAND_MAX;
//         while (pick < 0.0001)
//             pick = ((float) rand()) / RAND_MAX;

//         for (int j = 0; j < populationSize; j++) {
//             pick -= populationPro[j];
//             if (pick <= 0) {
//                 index[i] = j;
//                 break;
//             }
//         }
//     }

//     //是否精英选择
//     int elitismSize = populationSize;
//     if (elitism == TRUE) {
//         int *rank;
//         rank = rankForElitism();
//         elitismSize = (int) (populationSize - eliteCount);

//         //在新种群的最后保留eliteCount个个体
//         for (int i = elitismSize, k = 0; i < populationSize && k < eliteCount; i++, k++) {
//             for (int j = 0; j < chromosomeSize; j++) {
//                 tmpPopulationArray[i][j] = populationArray[rank[k]][j];
//             }
//             tmpFitness[i] = fitness[rank[k]];
//         }
//     }
//     for (int i = 0; i < elitismSize; i++) {
//         for (int j = 0; j < chromosomeSize; j++) {
//             tmpPopulationArray[i][j] = populationArray[index[i]][j];
//         }
//         tmpFitness[i] = fitness[index[i]];
//     }
//     free(index);

//     //产生新种群
//     for (int i = 0; i < populationSize; i++) {
//         for (int j = 0; j < chromosomeSize; j++) {
//             populationArray[i][j] = tmpPopulationArray[i][j];
//         }
//         fitness[i] = tmpFitness[i];
//     }
// }

// //cross function 每两个个体做判断
// __device__ void crossFcn() {
//     for (int i = 0; i < populationSize; i += 2) {
//         //判断当前两个个体是否做交叉
//         float pick1 = ((float) rand()) / RAND_MAX;
//         if (pick1 > crossRate)
//             continue;

//         for (int j = 0; j < chromosomeSize; j++) {
//             //判断两个个体中的染色体是否做交叉
//             int pick2 = rand();
//             if (pick2 & 1) {
//                 float tmp = populationArray[i][j];
//                 populationArray[i][j] = populationArray[i + 1][j];
//                 populationArray[i + 1][j] = tmp;
//             }
//         }
//     }
// }

// //mutation function
//  __device__ void mutationFcn() {
//     float scale = 0.5, shrink = 0.75;
//     for (int i = 0; i < populationSize; i++) {
//         scale -= scale * shrink * i / maxGeneration;

//         //判断当前个体是否变异
//         float pick1 = ((float) rand()) / RAND_MAX;
//         if (pick1 > mutationRate)
//             continue;

//         for (int j = 0; j < chromosomeSize; j++) {
//             //判断当前染色体是否变异
//             int pick2 = rand();
//             if (pick2 & 1) {
//                 float tmpChromosome;
//                 do {
//                     float pick3 = ((float) rand()) / RAND_MAX * 2 - 1;
//                     tmpChromosome = populationArray[i][j] + scale * (UB[j] - LB[j]) * pick3;
//                     //判断是否越界
//                 } while (tmpChromosome > UB[j] || tmpChromosome < LB[j]);
//                 populationArray[i][j] = tmpChromosome;
//             }
//         }
//     }
// }




int main(int argc, char *argv[]){
	time_t start = clock();
	srand(time(NULL));

	if(argc != 6){
		printf("ERROR\n");
		return 0;
	}

    BOOL success = TRUE;

    aRow = atoi(argv[2]);
    a = createMatrix(aRow, 16);
    success = getData(argv[1], a, aRow, 16);
    if (!success) {
        return 0;
    }
    aa = createMatrix(10, 9);
    success = getData(argv[3], aa, 10, 9);
    if (!success) {
        return 0;
    }

    aaaRow = atoi(argv[5]);
    aaa = createMatrix(aaaRow, 11);
    success = getData(argv[4], aaa, aaaRow, 11);
    if (!success) {
        return 0;
    }
    testPreData();


    hipMemcpyToSymbol(HIP_SYMBOL(c_a), a, aRow * 16 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aa), aa, 10 * 9 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aaa), aaa, aaaRow * 11 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aRow), &aRow, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aaaRow), &aaaRow, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_LB), LB, 10 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_UB), UB, 10 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_Dysum), Dysum, 9 * sizeof(float));

    
    float *populationArray;
    float *fitness;
    float *populationPro;
    float *X_10;
    fval = 100;
    //BOOL elitism = TRUE;

    hipMallocManaged(&populationArray, populationSize * chromosomeSize * sizeof(float));
    hipMallocManaged(&fitness, populationSize * sizeof(float));
    hipMallocManaged(&populationPro, populationSize * sizeof(float));
    hipMallocManaged(&X_10, 10 * sizeof(float));


    //hipMemset(Dzsum, 0, 9 * sizeof(float));
    hipMemset(populationArray, 0, populationSize * chromosomeSize * sizeof(float));
    hipMemset(fitness, 0, populationSize * sizeof(float));
    hipMemset(populationPro, 0, populationSize * sizeof(float));
    hipMemset(X_10, 0, 10 * sizeof(float));

    //initial population
    initial(populationArray);

    //fitness function
    GfitnessFcn<<<1, 100>>>(populationArray, fitness);
    hipDeviceSynchronize();
    // for (int i = 0; i < populationSize; i++) {
    //     float *x = (float*)malloc(chromosomeSize * sizeof(float));
    //     for (int j = 0; j < chromosomeSize; j++) {
    //         x[j] = populationArray[i * chromosomeSize + j];
    //     }
    //     fitness[i] = HfitnessFcn(x);
    //     free(x);
    // }

    

    hipFree(c_Dysum);
    hipFree(c_LB);
    hipFree(c_UB);
    hipFree(populationArray);
    hipFree(fitness);
    hipFree(populationPro);
    hipFree(X_10);

   
    time_t stop = clock();
    printf("time:%e\n", ((float) (stop - start)) / CLOCKS_PER_SEC);


    free(a);
    free(aa);
    free(aaa);

    return 0;
}