#include <hip/hip_runtime.h>
 
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <malloc.h>

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize



//typedef float float;
float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数
//BOOL elitism = TRUE; //是否精英选择

float *createMatrix(int rows, int cols) {
    float *matrix = (float*)malloc(rows * cols * sizeof(float));
    return matrix;
}


__global__ void initSeed(unsigned int seed, hiprandState_t* states) {
    int idx = threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

//Get data from files
BOOL getData(const char *fileName, float *x, int rows, int cols) {
    // open file to read data
    FILE *fp;
    fp = fopen(fileName, "r");
    if (fp == NULL) {
        printf("Open file %s error!!\n", fileName);
        return FALSE;
    }

    // read data
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fscanf(fp, "%f", (x + i * cols + j));
        }
    }
    return TRUE;
}

// //sig2ext in rainflow
__host__ __device__ float *sig2ext(float *sigy, float *dty, int rows, int *lenOfArray) {

    int n = 0, m = 0, k = 0, l = 0;

    //w=diff(sig);
    //w=logical([1;(w(1:end-1).*w(2:end))<=0;1]);
    float *w = (float*)malloc(rows * sizeof(float));
    for (int i = 1; i < rows; i++) {
        w[i - 1] = sigy[i] - sigy[i - 1];
    }

    for (int i = rows - 2; i > 0; i--) {
        float tmp = w[i] * w[i - 1];
        if (tmp <= 0) {
            w[i] = 1;
        } else {
            w[i] = 0;
            n++;
        }
    }
    w[0] = w[rows - 1] = 1;

    //ext=sigy(w);  exttime=dty(w);
    float *ext = (float*)malloc((rows - n) * sizeof(float));
    float *exttime = (float*)malloc((rows - n) * sizeof(float));
    for (int i = 0, j = 0; i < rows - n && j < rows;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = sigy[j];
            exttime[i] = dty[j];
            i++;
            j++;
        }
    }

    //w=diff(ext);
    //w=~logical([0; w(1:end-1)==0 & w(2:end)==0; 0]);
    for (int i = 1; i < rows - n; i++) {
        w[i - 1] = ext[i] - ext[i - 1];
    }

    for (int i = rows - n - 2; i > 0; i--) {
        if (w[i - 1] == 0 && w[i] == 0) {
            w[i] = 0;
            m++;
        } else {
            w[i] = 1;
        }
    }
    w[0] = w[rows - n - 1] = 1;

    //ext=ext(w); exttime=exttime(w);
    for (int i = 0, j = 0; i < rows - n - m && j < rows - n;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = ext[j];
            exttime[i] = exttime[j];
            i++;
            j++;
        }
    }

    //w=~logical([0; ext(1:end-1)==ext(2:end)]);
    for (int i = 1; i < rows - n - m; i++) {
        if (ext[i - 1] == ext[i]) {
            w[i] = 0;
            k++;
        } else {
            w[i] = 1;
        }
    }
    w[0] = 1;

    //ext=ext(w);
    for (int i = 0, j = 0; i < rows - n - m - k && j < rows - n - m;) {
        if (w[j] == 0) {
            j++;
        } else {
            ext[i] = ext[j];
            i++;
            j++;
        }
    }
    //w2=(exttime(2:end)-exttime(1:end-1))./2
    //exttime=[exttime(1:end-1)+w2.*~w(2:end); exttime(end)];
    //exttime=exttime(w);

    float *w2 = (float*)malloc((rows - n - m - 1) * sizeof(float));
    for (int i = 1; i < rows - n - m; i++) {
        w2[i - 1] = (exttime[i] - exttime[i - 1]) / 2.00;
    }

    for (int i = 0, j = 1; i < rows - n - m - 1 && j < rows - n - m;) {
        if (w[j] == 0) {
            exttime[i] = w2[i] * 1.00 + exttime[i];
            i++;
            j++;
        } else {
            exttime[i] = w2[i] * 0.00 + exttime[i];
            i++;
            j++;
        }
    }

    for (int i = 0, j = 0; i < rows - n - m - k && j < rows - n - m;) {
        if (w[j] == 0) {
            j++;
        } else {
            exttime[i] = exttime[j];
            i++;
            j++;
        }
    }

    //length(ext)>2,  w=diff(ext); w=logical([1; w(1:end-1).*w(2:end)<0; 1]);
    //ext4=ext(w); exttime=exttime(w);
    float *ext4 = NULL;
    *lenOfArray = 0;
    if (rows - n - m - k > 2) {
        for (int i = 1; i < rows - n - m - k; i++) {
            w[i - 1] = ext[i] - ext[i - 1];
        }

        for (int i = rows - n - m - k - 2; i > 0; i--) {
            if (w[i - 1] * w[i] < 0) {
                w[i] = 1;
            } else {
                w[i] = 0;
                l++;
            }
        }
        w[0] = 1;
        w[rows - n - m - k - 1] = 1;

        *lenOfArray = rows - n - m - k - l;
        ext4 = (float*)malloc(2 * (*lenOfArray) * sizeof(float));

        for (int i = 0, j = 0; i < rows - n - m - k - l && j < rows - n - m - k;) {
            if (w[j] == 0) {
                j++;
            } else {
                ext4[i] = ext[j];
                ext4[i + (*lenOfArray)] = exttime[j];
                i++;
                j++;
            }
        }
    }

    free(w);
    free(w2);
    free(ext);
    free(exttime);
    return ext4;
}

// //rainFlow in rainflow
__host__ __device__ float *rainFlow(float *ext, float *exttime, int lenOfSig2ext, int *lenOfRainflow) {
    float *rfy = NULL, *rfyResult = NULL;

    //function rfy5
    float a[128], t[128], ampl, mean, period, atime;
    int cNr = 1;
    int j = -1;

    //create 2D rfy(5 * (lenOfSig2ext -1))
    rfy = (float*)malloc(5 * (lenOfSig2ext - 1) * sizeof(float));

    int columnId = 0;
    int pointId = 0;

    for (int i = 0; i < lenOfSig2ext; i++) {
        a[++j] = *(ext + pointId);
        t[j] = *(exttime + pointId);
        while ((j >= 2) && (fabs(a[j - 1] - a[j - 2]) <= fabs(a[j] - a[j - 1]))) {
            ampl = fabs((a[j - 1] - a[j - 2]) / 2);
            switch (j) {
                case 0: {
                    break;
                }
                case 1: {
                    break;
                }
                case 2: {
                    mean = (a[0] + a[1]) / 2;
                    period = (t[1] - t[0]) * 2;
                    atime = t[0];
                    a[0] = a[1];
                    a[1] = a[2];
                    t[0] = t[1];
                    t[1] = t[2];
                    j = 1;
                    if (ampl > 0) {
                        *(rfy + columnId*5 + 0) = ampl;
                        *(rfy + columnId*5 + 1) = mean;
                        *(rfy + columnId*5 + 2) = 0.50;
                        *(rfy + columnId*5 + 3) = atime;
                        *(rfy + columnId*5 + 4) = period;
                        columnId++;
                    }
                    break;
                }
                default: {
                    mean = (a[j - 1] + a[j - 2]) / 2;
                    period = (t[j - 1] - t[j - 2]) * 2;
                    atime = t[j - 2];
                    a[j - 2] = a[j];
                    t[j - 2] = t[j];
                    j = j - 2;
                    if (ampl > 0) {
                        *(rfy + columnId*5 + 0) = ampl;
                        *(rfy + columnId*5 + 1) = mean;
                        *(rfy + columnId*5 + 2) = 1.00;
                        *(rfy + columnId*5 + 3) = atime;
                        *(rfy + columnId*5 + 4) = period;
                        columnId++;
                        cNr++;
                    }
                    break;
                }
            }
        }
        pointId++;
    }
    for (int i = 0; i < j; i++) {
        ampl = fabs(a[i] - a[i + 1]) / 2;
        mean = (a[i] + a[i + 1]) / 2;
        period = (t[i + 1] - t[i]) * 2;
        atime = t[i];
        if (ampl > 0) {
            *(rfy + columnId*5 + 0) = ampl;
            *(rfy + columnId*5 + 1) = mean;
            *(rfy + columnId*5 + 2) = 0.50;
            *(rfy + columnId*5 + 3) = atime;
            *(rfy + columnId*5 + 4) = period;
            columnId++;
        }
    }

    //create 2D rfyResult(5 * (lenOfSig2ext - cNr))
    rfyResult = (float*)malloc(5 * (lenOfSig2ext - cNr) * sizeof(float));

    *lenOfRainflow = lenOfSig2ext - cNr;

    for (int i = 0; i < 5 * (lenOfSig2ext - cNr); i++) {
            rfyResult[i] = rfy[i];
    }

    free(rfy);
    return rfyResult;
}

// //rfhist in rainflow
__host__ __device__ float *rfhist(float *rfy, int lenOfRainflow, int *lenOfRfhist) {
    float *noy = NULL, *xoy = NULL;
    int x = 32;
    *lenOfRfhist = x;

    xoy = (float*)malloc(x * sizeof(float));
    noy = (float*)malloc(x * sizeof(float));
    memset(noy, 0, x * sizeof(float));

    //halfc=find(rfy(3,:)==0.5)
    int halfcNum = 0;
    for (int i = 0; i < lenOfRainflow; i++) {
        if (rfy[i * 5 + 2] == 0.50)
            halfcNum++;
    }

    int *halfc = NULL;
    halfc = (int*)malloc(halfcNum * sizeof(int));
    for (int i = 0, j = 0; i < lenOfRainflow && j < halfcNum;) {
        if (rfy[i * 5 + 2] == 0.50) {
            halfc[j] = i;
            j++;
        }
        i++;
    }

    float min = rfy[0], max = rfy[0];
    for (int i = 0; i < lenOfRainflow; i++) {
        if (rfy[i * 5] >= max) {
            max = rfy[i * 5];
        } else if (rfy[i * 5] <= min) {
            min = rfy[i * 5];
        }
    }

    float wid = (max - min) / x;
    for (int i = 0; i < x; i++) {
        xoy[i] = min + (float) (i + 0.50) * wid;
    }

    for (int i = 0; i < lenOfRainflow; i++) {
        int j;
        j = (int) floor((rfy[i * 5] - min) / wid);
        if (j != 0 && fabs((rfy[i * 5] - min) - wid * j) < 1e-10) {
            noy[j - 1] += 1;
        } else {
            noy[j] += 1;
        }
    }

    //if ~isempty(halfc) {
    //[N2 x]=hist(rf(r,halfc),x)  N2 = noy2, x = *xoy
    //N1=N1-0.5*N2  N1 = noy
    // }
    if (halfcNum != 0) {
        float *noy2 = (float*)malloc(x * sizeof(float));
        memset(noy2, 0, x * sizeof(float));
        float *rf = (float*)malloc(halfcNum * sizeof(float));
        for (int i = 0; i < halfcNum; i++) {
            int j = halfc[i];
            rf[i] = rfy[j * 5];
        }

        for (int i = 0; i < halfcNum; i++) {
            int j;
            j = (int) floor((rf[i] - min) / wid);
            if (j != 0 && fabs((rf[i] - min) - wid * j) < 1e-10) {
                noy2[j - 1] += 1;
            } else {
                noy2[j] += 1;
            }
        }

        for (int i = 0; i < x; i++) {
            noy[i] -= noy2[i] * 0.5;
        }

        free(noy2);
        free(rf);
    }

    float *rfhist = (float*)malloc(2 * x * sizeof(float));
    for (int i = 0; i < x; i++) {
        rfhist[i] = noy[i];
        rfhist[i + x] = xoy[i];
    }

    free(halfc);
    free(xoy);
    free(noy);

    return rfhist;
}

void testPreData() {
    for (int kk = 0; kk < 9; kk++) {
        float *sigy = (float*)malloc(aaaRow * sizeof(float));
        float *dty = (float*)malloc(aaaRow * sizeof(float));

        for (int i = 0; i < aaaRow; i++) {
            sigy[i] = aaa[i * 11 + kk + 2];
            dty[i] = aaa[i * 11 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sigy, dty, aaaRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rfy = NULL;
        int lenOfRainflow;
        rfy = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *noy = NULL, *xoy = NULL;
        int lenOfRfhist;
        noy = rfhist(rfy, lenOfRainflow, &lenOfRfhist);
        xoy = noy + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dysum[kk] += noy[i] * pow(xoy[i] * 0.21 / 70, 3.5);
        }
        //printf("%e\n", Dysum[kk]);

        free(sigy);
        free(dty);
        free(ext);
        free(rfy);
        free(noy);
    }
}

 //fitness Function
float HfitnessFcn(float *x) {
    //initial Dzsum in every generation
    float *Dzsum = (float*)malloc(9 * sizeof(float));
    memset(Dzsum, 0, sizeof(float) * 9);

    float *Tzb = (float*)malloc(aRow * 9 * sizeof(float));
    memset(Tzb, 0, aRow * 9 * sizeof(float));

    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < aRow; j++) {
            Tzb[j * 9 + i] = x[0] * aa[0 * 9 + i] * a[j * 16 + 2] + x[1] * aa[1 * 9 + i] * a[j * 16 + 3] + x[2] * aa[2 *0 + i] * a[j * 16 + 4] +
                            x[3] * aa[3 * 9 + i] * a[j * 16 + 5] + x[4] * aa[4 * 9 + i] * a[j * 16 + 6] + x[5] * aa[5 * 9 + i] * a[j * 16 + 7] +
                            x[6] * aa[6 * 9 + i] * a[j * 16 + 8] + x[7] * aa[7 * 9 + i] * a[j * 16 + 9] + x[8] * aa[8 * 9 + i] * a[j * 16 + 10] +
                            x[9] * aa[9 * 9 + i] * a[j * 16 + 11];
        }
    }

    for (int k = 0; k < 9; k++) {
        float *sig = (float*)malloc(aRow * sizeof(float));
        float *dt = (float*)malloc(aRow * sizeof(float));
        for (int i = 0; i < aRow; i++) {
            sig[i] = Tzb[i * 9 + k];
            dt[i] = a[i * 9 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sig, dt, aRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rf = NULL;
        int lenOfRainflow;
        rf = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *no = NULL, *xo = NULL;
        int lenOfRfhist;
        no = rfhist(rf, lenOfRainflow, &lenOfRfhist);
        xo = no + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dzsum[k] += no[i] * pow(xo[i] * 0.21 / 70, 3.5);
        }
        //printf("%e\n", Dzsum[k]);

        free(sig);
        free(dt);
        free(ext);
        free(rf);
        free(no);
    }

    float y = 0;
    for (int i = 0; i < 9; i++) {
        //constraint : c =(Dysum[i]-Dzsum[i]) <= 0
        float c = Dysum[i] - Dzsum[i];
        if (c <= 0) {
            y += pow(c, 2);
        } else {
            y = 100;
        }
    }
    //printf("%e\n", y);

    free(Dzsum);
    free(Tzb);

    return y;
}

void initial(float *populationArray){
    for (int i = 0; i < populationSize; i++) {
        float *x = (float*)malloc(chromosomeSize * sizeof(float));
        for (int j = 0; j < chromosomeSize; j++) {
            int high_pos = rand();
            int low_pos = (rand() & ((1 << 16) - 1));
            high_pos = (high_pos & ((1 << 15) - 1));
            int value = low_pos + (high_pos << 16);
            populationArray[i * chromosomeSize + j] = (UB[j] - LB[j]) * ((float) value / ((1U << 31) - 1)) + LB[j];
            x[j] = populationArray[i * chromosomeSize + j];
        }
        float tmp_fit = HfitnessFcn(x);
        if (tmp_fit > 99) {
            i--;
        }
        //printf("%e\n",tmp_fit);
        free(x);
    }
}

__device__ float DfitnessFcn(float *x) {
    //initial Dzsum in every generation
    float *Dzsum = (float*)malloc(9 * sizeof(float));
    memset(Dzsum, 0, sizeof(float) * 9);

    float *Tzb = (float*)malloc(c_aRow * 9 * sizeof(float));
    memset(Tzb, 0, c_aRow * 9 * sizeof(float));


    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < c_aRow; j++) {
            Tzb[j * 9 + i] = x[0] * c_aa[0 * 9 + i] * c_a[j * 16 + 2] + x[1] * c_aa[1 * 9 + i] * c_a[j * 16 + 3] + x[2] * c_aa[2 *0 + i] * c_a[j * 16 + 4] +
                            x[3] * c_aa[3 * 9 + i] * c_a[j * 16 + 5] + x[4] * c_aa[4 * 9 + i] * c_a[j * 16 + 6] + x[5] * c_aa[5 * 9 + i] * c_a[j * 16 + 7] +
                            x[6] * c_aa[6 * 9 + i] * c_a[j * 16 + 8] + x[7] * c_aa[7 * 9 + i] * c_a[j * 16 + 9] + x[8] * c_aa[8 * 9 + i] * c_a[j * 16 + 10] +
                            x[9] * c_aa[9 * 9 + i] * c_a[j * 16 + 11];
        }
    }

    for (int k = 0; k < 9; k++) {
        float *sig = (float*)malloc(c_aRow * sizeof(float));
        float *dt = (float*)malloc(c_aRow * sizeof(float));
        for (int i = 0; i < c_aRow; i++) {
            sig[i] = Tzb[i * 9 + k];
            dt[i] = c_a[i * 9 + 1];
        }

        float *ext = NULL, *exttime = NULL;
        int lenOfSig2ext;
        ext = sig2ext(sig, dt, c_aRow, &lenOfSig2ext);
        exttime = ext + lenOfSig2ext;

        float *rf = NULL;
        int lenOfRainflow;
        rf = rainFlow(ext, exttime, lenOfSig2ext, &lenOfRainflow);

        float *no = NULL, *xo = NULL;
        int lenOfRfhist;
        no = rfhist(rf, lenOfRainflow, &lenOfRfhist);
        xo = no + lenOfRfhist;

        for (int i = 0; i < lenOfRfhist; i++) {
            Dzsum[k] += no[i] * pow(xo[i] * 0.21 / 70, 3.5);
        }
        printf("%e\n", Dzsum[k]);

        free(sig);
        free(dt);
        free(ext);
        free(rf);
        free(no);
    }

    float y = 0;
    for (int i = 0; i < 9; i++) {
        //constraint : c =(Dysum[i]-Dzsum[i]) <= 0
        float c = c_Dysum[i] - Dzsum[i];
        if (c <= 0) {
            y += pow(c, 2);
        } else {
            y = 100;
        }
    }
    printf("%e\n", y);

    free(Dzsum);
    free(Tzb);

    return y;
}

__global__ void GfitnessFcn(float *populationArray, float *fitness){
    int idx = blockIdx.x + threadIdx.x;
    float *x = (float*)malloc(chromosomeSize * sizeof(float));
    memset(x, 0, chromosomeSize * sizeof(float));
    for (int j = 0; j < chromosomeSize; j++) {
        *(x + j) = *(populationArray +idx * chromosomeSize + j);
    }
    fitness[idx] = DfitnessFcn(x);
    free(x);
}

//sum fitness
__host__ __device__ float sum(float *x) {
    float sum = 0;
    for (int i = 0; i < populationSize; i++) {
        sum += x[i];
    }
    return sum;
}

//best fitness position
float *bestFitness(float *fitness) {

    //bestRes[bestFitness][bestIndex]
    float bestFitness = fitness[0];
    int bestIndex = 0;
    float *bestRes = (float*)malloc(2 * sizeof(float));
    for (int i = 0; i < populationSize; i++) {
        if (fitness[i] < bestFitness) {
            bestFitness = fitness[i];
            bestIndex = i;
        }
    }
    bestRes[0] = bestFitness;
    bestRes[1] = bestIndex;

    return bestRes;
}

//select pre
__global__ void selectPre(float *fitness, float *Fitness, float *tmpFitness, float *populationArray, float *tmpPopulationArray){
    int idx = threadIdx.x;
    Fitness[idx] = 1 / fitness[idx];
    __syncthreads();
    tmpFitness[idx] = fitness[idx];
    __syncthreads();
    for(int i = 0; i < chromosomeSize; i++){
        tmpPopulationArray[idx * chromosomeSize + i] = populationArray[idx * chromosomeSize + i];
    }
    __syncthreads();
}

//select function 轮盘选择
__global__ void selectFcn(float *populationArray, float *tmpPopulationArray, float *fitness, float *Fitness, float *tmpFitness, float sumFitness, float *populationPro, hiprandState_t *states) {
    //printf("selectFcn\n");
    int idx = threadIdx.x;

    //每个个体被选择的概率
    populationPro[idx] = Fitness[idx] / sumFitness;
    __syncthreads();

    //轮盘选择
    int index;
        hiprandState_t s;
        s = states[idx];
        float ss = hiprand_uniform(&s);
        while (ss < 0.0001)
            ss = hiprand_uniform(&s);
            //printf("%e\n", ss);

        for (int j = 0; j < populationSize; j++) {
            ss -= populationPro[j];
            if (ss <= 0) {
                index = j;
                //printf("%d\n", index);
                break;
            }
        }

    //产生新种群
    for (int j = 0; j < chromosomeSize; j++) {
        populationArray[idx * chromosomeSize + j] = tmpPopulationArray[index * chromosomeSize + j];
    }
    __syncthreads();
    fitness[idx] = tmpFitness[index];
    __syncthreads();
}

//cross function 每两个个体做判断
__global__ void crossFcn(float *populationArray, hiprandState_t *states) {
    //printf("crossFcn\n");

    int idx = threadIdx.x;
    hiprandState_t s = states[idx];
    hiprandState_t t = states[idx];
    float ss = hiprand_uniform(&s);
    int tt = hiprand(&t);


    //判断当前两个个体是否做交叉
    if (ss < crossRate){
       for (int j = 0; j < chromosomeSize; j++) {
        //判断两个个体中的染色体是否做交叉
            if (tt % 2 != 0) {
                float tmp = populationArray[idx * chromosomeSize + j];
                populationArray[idx * chromosomeSize + j] = populationArray[(idx + populationSize/2) * chromosomeSize + j];
                populationArray[(idx + populationSize/2) * chromosomeSize + j] = tmp;
            }
        }
    }    
}

//mutation function
__global__ void mutationFcn(float *populationArray, hiprandState_t *states) {
    //printf("mutationFcn\n");
    int idx = threadIdx.x;
    hiprandState_t s = states[idx];
    hiprandState_t t = states[idx];
    float ss = hiprand_uniform(&s);
    int tt = hiprand(&t);

    float scale = 0.5, shrink = 0.75;
    scale -= scale * shrink * idx / maxGeneration;

    //判断当前个体是否变异
    if (ss < mutationRate){
        for (int j = 0; j < chromosomeSize; j++) {
            //判断当前染色体是否变异
            if (tt % 2 != 0) {
                float tmpChromosome;
                do {
                    tmpChromosome = populationArray[idx * chromosomeSize + j] + scale * (c_UB[j] - c_LB[j]) * ss;
                    //判断是否越界
                } while (tmpChromosome > c_UB[j] || tmpChromosome < c_LB[j]);
                populationArray[idx * chromosomeSize + j] = tmpChromosome;
            }
        }   
    }    
}


//rank fitness
int *rankForElitism(float *fitness) {

    // initialize rank array
    int *rank = (int *)malloc(populationSize * sizeof(int));
    for (int i = 0; i < populationSize; i++) {
        rank[i] = i;
    }

    // rank fitness in increase order
    for (int i = populationSize - 1; i > 0; i--) {
        for (int j = 0; j < i; j++) {
            if (fitness[rank[j]] > fitness[rank[j + 1]]) {
                int tmp_rank = rank[j];
                rank[j] = rank[j + 1];
                rank[j + 1] = tmp_rank;
            }
        }
    }

    return rank;
}

//select function 轮盘选择
void selectFcn(float *populationArray, float *fitness, float *populationPro) {

    // float tmpPopulationArray[populationSize * chromosomeSize];
    // float tmpFitness[populationSize];

    float *tmpPopulationArray = (float*)malloc(populationSize * chromosomeSize * sizeof(float));
    float *tmpFitness = (float*)malloc(populationSize * sizeof(float));

    for (int i = 0; i < populationSize; i++) {
        for (int j = 0; j < chromosomeSize; j++) {
            tmpPopulationArray[i * chromosomeSize + j] = populationArray[i * chromosomeSize + j];
        }
        tmpFitness[i] = fitness[i];
    }

    //每个个体被选择的概率
    float *Fitness = (float*)malloc(populationSize * sizeof(float));
    float sumFitness = 0;

    for (int i = 0; i < populationSize; i++) {
        Fitness[i] = 1 / fitness[i];
    }

    sumFitness = sum(Fitness);
    for (int i = 0; i < populationSize; i++) {
        populationPro[i] = Fitness[i] / sumFitness;
    }
    free(Fitness);

    //轮盘选择
    int *index = (int*)malloc(populationSize * sizeof(int));
    for (int i = 0; i < populationSize; i++) {
        float pick = ((float) rand()) / RAND_MAX;
        while (pick < 0.0001)
            pick = ((float) rand()) / RAND_MAX;

        for (int j = 0; j < populationSize; j++) {
            pick -= populationPro[j];
            if (pick <= 0) {
                index[i] = j;
                //printf("%d\n", index[i]);
                break;
            }
        }
    }

    //产生新种群
    for (int i = 0; i < populationSize; i++) {
        for (int j = 0; j < chromosomeSize; j++) {
            populationArray[i * chromosomeSize + j] = tmpPopulationArray[index[i] * chromosomeSize + j];
        }
        fitness[i] = tmpFitness[index[i]];
    }
    free(index);
    free(tmpPopulationArray);
    free(tmpFitness);
}


//cross function 每两个个体做判断
void crossFcn(float *populationArray) {
    //printf("crossFcn\n");
    for (int i = 0; i < populationSize; i += 2) {
        //判断当前两个个体是否做交叉
        float pick1 = ((float) rand()) / RAND_MAX;
        if (pick1 > crossRate)
            continue;

        for (int j = 0; j < chromosomeSize; j++) {
            //判断两个个体中的染色体是否做交叉
            int pick2 = rand();
            if (pick2 & 1) {
                float tmp = populationArray[i * chromosomeSize + j];
                populationArray[i * chromosomeSize + j] = populationArray[(i+1) * chromosomeSize + j];
                populationArray[(i+1) * chromosomeSize + j] = tmp;
            }
        }
    }
}

//mutation function
void mutationFcn(float *populationArray) {
    //printf("mutationFcn\n");
    float scale = 0.5, shrink = 0.75;
    for (int i = 0; i < populationSize; i++) {
        scale -= scale * shrink * i / maxGeneration;

        //判断当前个体是否变异
        float pick1 = ((float) rand()) / RAND_MAX;
        if (pick1 > mutationRate)
            continue;

        for (int j = 0; j < chromosomeSize; j++) {
            //判断当前染色体是否变异
            int pick2 = rand();
            if (pick2 & 1) {
                float tmpChromosome;
                do {
                    float pick3 = ((float) rand()) / RAND_MAX * 2 - 1;
                    tmpChromosome = populationArray[i * chromosomeSize + j] + scale * (UB[j] - LB[j]) * pick3;
                    //判断是否越界
                } while (tmpChromosome > UB[j] || tmpChromosome < LB[j]);
                populationArray[i * chromosomeSize + j] = tmpChromosome;
            }
        }
    }
}





int main(int argc, char *argv[]){
	time_t start = clock();
	srand(time(NULL));

	if(argc != 6){
		printf("ERROR\n");
		return 0;
	}

    BOOL success = TRUE;

    aRow = atoi(argv[2]);
    a = createMatrix(aRow, 16);
    success = getData(argv[1], a, aRow, 16);
    if (!success) {
        return 0;
    }
    aa = createMatrix(10, 9);
    success = getData(argv[3], aa, 10, 9);
    if (!success) {
        return 0;
    }

    aaaRow = atoi(argv[5]);
    aaa = createMatrix(aaaRow, 11);
    success = getData(argv[4], aaa, aaaRow, 11);
    if (!success) {
        return 0;
    }
    testPreData();


    hipMemcpyToSymbol(HIP_SYMBOL(c_a), a, aRow * 16 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aa), aa, 10 * 9 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aaa), aaa, aaaRow * 11 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aRow), &aRow, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_aaaRow), &aaaRow, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_LB), LB, 10 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_UB), UB, 10 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_Dysum), Dysum, 9 * sizeof(float));

    
    float *populationArray;
    float *fitness;
    float *populationPro;
    float *Fitness;
    float *tmpPopulationArray;
    float *tmpFitness;
    float *X_10;
    fval = 100;
    //BOOL elitism = TRUE;

    hipMallocManaged(&populationArray, populationSize * chromosomeSize * sizeof(float));
    hipMallocManaged(&fitness, populationSize * sizeof(float));
    hipMallocManaged(&populationPro, populationSize * sizeof(float));
    hipMallocManaged(&Fitness, populationSize * sizeof(float));
    hipMallocManaged(&tmpPopulationArray, populationSize * chromosomeSize * sizeof(float));
    hipMallocManaged(&tmpFitness, populationSize * sizeof(float));
    hipMallocManaged(&X_10, 10 * sizeof(float));

    hipMemset(populationArray, 0, populationSize * chromosomeSize * sizeof(float));
    hipMemset(fitness, 0, populationSize * sizeof(float));
    hipMemset(populationPro, 0, populationSize * sizeof(float));
    hipMemset(Fitness, 0, populationSize * sizeof(float));
    hipMemset(tmpPopulationArray, 0, populationSize * chromosomeSize * sizeof(float));
    hipMemset(tmpFitness, 0, populationSize * sizeof(float));
    hipMemset(X_10, 0, 10 * sizeof(float));


    hiprandState_t *states = NULL;
    hipMalloc((void**)&states, populationSize * sizeof(hiprandState_t));



    initSeed<<<1, populationSize>>>(time(NULL), states);
    hipDeviceSynchronize();

    //initial population
    initial(populationArray);

    for(int n = 0; n < maxGeneration; n++){

        for (int i = 0; i < populationSize; i++) {
            float *x = (float*)malloc(chromosomeSize * sizeof(float));
            for (int j = 0; j < chromosomeSize; j++) {
                x[j] = populationArray[i * chromosomeSize + j];
            }
            fitness[i] = HfitnessFcn(x);
            free(x);
        }


        //每一代最优适应度及其位置
        //bestRes[bestFitness][bestIndex]
        float *bestRes = bestFitness(fitness);
        bestFitnessOfGen = bestRes[0];
        bestIndexOfGen = (int) bestRes[1];

        if (bestFitnessOfGen < fval) {
            fval = bestFitnessOfGen;
            for (int k = 0; k < chromosomeSize; k++) {
                X_10[k] = populationArray[bestIndexOfGen * chromosomeSize + k];
            }
            G = n + 1;
        }
        // printf("1bestFitness : %e\n", bestFitnessOfGen);
        // printf("1fval : %e\n", fval);
        
        //printf("%e\n", bestFitnessOfGen);
        //printf("%d\n", bestIndexOfGen);

        free(bestRes);
        if(G == maxGeneration - 1) break;


        selectPre<<<1, populationSize>>>(fitness, Fitness, tmpFitness, populationArray, tmpPopulationArray);
        hipDeviceSynchronize();

        float sumFitness = sum(Fitness);

        selectFcn<<<1, populationSize>>>(populationArray, tmpPopulationArray, fitness, Fitness, tmpFitness, sumFitness, populationPro, states);
        hipDeviceSynchronize();
        //selectFcn(populationArray, fitness, populationPro);

        crossFcn<<<1, populationSize/2>>>(populationArray, states);    
        hipDeviceSynchronize();
        //crossFcn(populationArray);

        mutationFcn<<<1, populationSize>>>(populationArray, states);
        hipDeviceSynchronize();
        //mutationFcn(populationArray);

    }


    printf("fval:%e\n", fval);
    printf("X:%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", X_10[0], X_10[1], X_10[2], X_10[3], X_10[4], X_10[5], X_10[6],
           X_10[7], X_10[8], X_10[9]);
    printf("Gen:%d\n", G);
   
    time_t stop = clock();
    printf("time:%e\n", ((float) (stop - start)) / CLOCKS_PER_SEC);


    hipFree(c_Dysum);
    hipFree(c_LB);
    hipFree(c_UB);
    hipFree(populationArray);
    hipFree(fitness);
    hipFree(populationPro);
    hipFree(Fitness);
    hipFree(tmpPopulationArray);
    hipFree(tmpFitness);
    hipFree(X_10);
    hipFree(states);


    free(a);
    free(aa);
    free(aaa);

    return 0;
}